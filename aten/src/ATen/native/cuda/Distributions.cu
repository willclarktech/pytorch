#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGenerator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

/**
 * Note [Register spilling in hiprand call for CUDA < 10]
 * ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
 * For CUDA < 10, hiprandStatePhilox4_32_10_t engine achieves poor performance (60% SOL bandwidth)
 * when called to generate one random number at a time. This is because the line
 *            unsigned ret = (&state->output.x)[state->STATE++];
 * in
 *            QUALIFIERS unsigned int hiprand(hiprandStatePhilox4_32_10_t *state)
 * in hiprand/hiprand_kernel.h dynamically indexes into state.output, preventing the compiler from ever
 * storing state.output in registers.
 *
 * CUDA 10 fixed this problem. However, for backwards compatibility, in the following kernels
 * we are using hiprand distributions that utilize curand4 call. curand4 call doesn't have the
 * register spilling problem.
 */

namespace {

template <typename scalar_t>
void poisson_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& lambda,
    std::pair<uint64_t, uint64_t> seeds) {
  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t>(
      ret,
      lambda,
      [seeds] __device__(
          scalar_t & ret_val, const scalar_t& lambda) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);
        ret_val = static_cast<scalar_t>(hiprand_poisson(&state, lambda));
      });
}

template <typename scalar_t>
void gamma_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& alpha,
    std::pair<uint64_t, uint64_t> seeds) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t>(
      ret,
      alpha,
      [seeds] __device__(
          scalar_t & ret_val, const scalar_t& alpha) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);

        auto uniform_lambda = [&state] __device__ () {
          return hiprand_uniform(&state);
        };
        BaseSampler<accscalar_t, decltype(uniform_lambda)> standard_uniform(uniform_lambda);

        auto normal_lambda = [&state] __device__ () {
          return hiprand_normal(&state);
        };
        BaseSampler<accscalar_t, decltype(normal_lambda)> standard_normal(normal_lambda);
        auto sample = sample_gamma<scalar_t, accscalar_t, decltype(uniform_lambda), decltype(normal_lambda)>(alpha, standard_uniform, standard_normal);
        auto min_value = std::numeric_limits<scalar_t>::min();
        ret_val = (min_value > sample) ? min_value : sample;
      });
}

template <typename scalar_t>
void gamma_grad_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& self,
    const at::Tensor& output) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, scalar_t>(
      ret, self, output,
      [] __device__ (scalar_t& ret_val, const scalar_t& self_val, const scalar_t &output_val) {
        ret_val = standard_gamma_grad_one<scalar_t, accscalar_t>(self_val, output_val);
      });
}

template <typename scalar_t>
void dirichlet_grad_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& x,
    const at::Tensor& alpha,
    const at::Tensor& total) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::cuda::CUDA_tensor_apply4<scalar_t, scalar_t, scalar_t, scalar_t>(
      ret, x, alpha, total,
      [] __device__ (scalar_t& ret_val, const scalar_t& x_val, const scalar_t& alpha_val, const scalar_t& total_val) {
        ret_val = dirichlet_grad_one<scalar_t, accscalar_t>(x_val, alpha_val, total_val);
      });
}

template<typename scalar_t, typename prob_t>
void bernoulli_tensor_cuda_kernel(
    at::Tensor& ret, const at::Tensor& p,
    std::pair<uint64_t, uint64_t> seeds) {
  // The template argument `4` below indicates that we want to operate on four
  // element at each time. See NOTE [ CUDA_tensor_applyN helpers ] for details.
  at::cuda::CUDA_tensor_apply2<scalar_t, prob_t, 4>(
      ret, p,
      [seeds] __device__(
          int n, scalar_t& v1, scalar_t& v2, scalar_t& v3, scalar_t& v4,
          const prob_t& p1, const prob_t& p2, const prob_t& p3, const prob_t& p4) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);
        // See Note [Register spilling in hiprand call for CUDA < 10]
        float4 rand = hiprand_uniform4(&state);
        switch (n) {
          case 4: {
            CUDA_KERNEL_ASSERT(0 <= p4 && p4 <= 1);
            v4 = static_cast<scalar_t>(rand.w <= p4);
            // fallthrough
          }
          case 3: {
            CUDA_KERNEL_ASSERT(0 <= p3 && p3 <= 1);
            v3 = static_cast<scalar_t>(rand.z <= p3);
            // fallthrough
          }
          case 2: {
            CUDA_KERNEL_ASSERT(0 <= p2 && p2 <= 1);
            v2 = static_cast<scalar_t>(rand.y <= p2);
            // fallthrough
          }
          case 1: {
            CUDA_KERNEL_ASSERT(0 <= p1 && p1 <= 1);
            v1 = static_cast<scalar_t>(rand.x <= p1);
          }
        }
      }
    );
}

template<typename scalar_t>
void dirichlet_scalar_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& gamma) {
  auto gamma_sum = gamma.sum(-1, true).expand(ret.sizes());
  at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, scalar_t>(ret, gamma, gamma_sum,
  [] __device__(scalar_t &ret_val, const scalar_t &gamma, const scalar_t &gamma_sum) {
    ret_val = gamma / gamma_sum;
    auto min_value = std::numeric_limits<scalar_t>::min();
    auto max_value = 1 - std::numeric_limits<scalar_t>::epsilon();
    ret_val = (min_value > ret_val) ? min_value : ret_val;
    ret_val = (max_value < ret_val) ? max_value : ret_val;
  });
}

} // namespace

namespace at { namespace native {

Tensor _s_poisson_cuda(const Tensor& lambda, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(20);
  }
  Tensor ret = at::empty(lambda.sizes(), lambda.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "poisson_cuda", [&] {
    poisson_cuda_kernel<scalar_t>(ret, lambda, rng_engine_inputs);
  });
  return ret;
}

Tensor _s_gamma_cuda(const Tensor& alpha, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(10);
  }
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "gamma_cuda", [&] {
     gamma_cuda_kernel<scalar_t>(ret, alpha, rng_engine_inputs);
   });
  return ret;
}

Tensor _s_dirichlet_cuda(const Tensor& alpha, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(10);
  }
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "dirichlet", [&] {
    Tensor gamma = at::empty(alpha.sizes(), alpha.options());
    gamma_cuda_kernel<scalar_t>(gamma, alpha, rng_engine_inputs);
    dirichlet_scalar_cuda_kernel<scalar_t>(ret, gamma);
  });
  return ret;
}

Tensor _standard_gamma_grad_cuda(const Tensor& self, const Tensor& output) {
  Tensor ret = at::empty(self.sizes(), self.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "_standard_gamma_grad_cuda", [&] {
     gamma_grad_cuda_kernel<scalar_t>(ret, self, output);
   });
  return ret;
}

Tensor _dirichlet_grad_cuda(const Tensor& x, const Tensor& alpha, const Tensor& total) {
  Tensor ret = at::empty(x.sizes(), x.options());
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "_dirichlet_grad_cuda", [&] {
    dirichlet_grad_cuda_kernel<scalar_t>(ret, x, alpha, total);
  });
  return ret;
}

Tensor& bernoulli_tensor_cuda_(Tensor &self, const Tensor& p_, Generator* gen_) {
  NoNamesGuard guard;
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(10);
  }
  auto p = std::get<0>(expand_inplace(self, p_.to(kCUDA)));
  AT_DISPATCH_ALL_TYPES_AND3(
    at::ScalarType::Half, at::ScalarType::BFloat16, at::ScalarType::Bool, self.scalar_type(), "bernoulli_tensor_cuda_self_", [&] {
      using self_t = scalar_t;
      AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, p.scalar_type(), "bernoulli_tensor_cuda_p_", [&] {
        using p_t = scalar_t;
        return bernoulli_tensor_cuda_kernel<self_t, p_t>(self, p, rng_engine_inputs);
      });
   });
  return self;
}

void uniform_kernel_cuda(TensorIterator& iter, double from_, double to_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "uniform_cuda", [&] {
    auto from = static_cast<scalar_t>(from_);
    auto to = static_cast<scalar_t>(to_);
    TORCH_CHECK(from <= to,
      "uniform_ expects to return a [from, to) range, but found from=", from,
      " > to=", to);
    TORCH_CHECK((to - from) <= std::numeric_limits<scalar_t>::max(),
          "uniform_ expects to-from <= std::numeric_limits<", toString(iter.dtype()),
          ">::max(), but found to=", to, " and from=", from,
          " which result in to-from to exceed the limit");

    using accscalar_t = at::acc_type<scalar_t, true>;
    auto range = static_cast<accscalar_t>(to-from);
    from = static_cast<accscalar_t>(from);
    // define lambda to reverse bounds, multiply 'range' and add 'from_'
    auto uniform_func = [range, from] __device__ (accscalar_t rand) {
      // reverse the bounds of curand4 from (0, 1] to [0, 1)
      // Note that this method is from legacy THCTensorRandom and is likely to give
      // you more 0-s, since, the probability of gettings 1-s is higher than 0-s and
      // by reversing the bounds, we are flipping the probabilities of 1-s and 0-s.
      auto reverse_bound_rand = rand == static_cast<accscalar_t>(1.0) ? static_cast<accscalar_t>(0.0) : rand;
      return static_cast<scalar_t>(reverse_bound_rand * range + from);
    };
    if (std::is_same<scalar_t, double>::value) {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        uniform_func);
    } else {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        uniform_func);
    }
   });
}

void random_from_to_kernel(TensorIterator& iter, uint64_t range, int64_t base, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  at::native::templates::cuda::random_from_to_kernel(iter, range, base, gen);
}

void random_full_64_bits_range_kernel(TensorIterator& iter, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  at::native::templates::cuda::random_full_64_bits_range_kernel(iter, gen);
}

void random_kernel(TensorIterator& iter, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  at::native::templates::cuda::random_kernel(iter, gen);
}

void normal_kernel_cuda(TensorIterator& iter, double mean_, double std_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "normal_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto mean = static_cast<accscalar_t>(mean_);
    auto std = static_cast<accscalar_t>(std_);
    // define lambda to multiply std and add mean
    auto normal_func = [mean, std] __device__ (accscalar_t rand) {
      return static_cast<scalar_t>(rand * std + mean);
    };
    if (std::is_same<scalar_t, double>::value) {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal2_double(state); },
        normal_func);
    } else {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal4(state); },
        normal_func);
    }
   });
}

void cauchy_kernel(TensorIterator& iter, double median_, double sigma_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "cauchy_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto median = static_cast<accscalar_t>(median_);
    auto sigma = static_cast<accscalar_t>(sigma_);
    if (std::is_same<scalar_t, double>::value) {
      // define lambda for cauchy transformation
      auto cauchy_func = [median, sigma] __device__ (accscalar_t rand) {
        return static_cast<scalar_t>(median + sigma *
                ::tan(static_cast<accscalar_t>(M_PI) * (rand-static_cast<accscalar_t>(0.5))));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        cauchy_func);
    } else {
      // use __tanf fast approximation for peak bandwidth
      auto cauchy_func = [median, sigma] __device__ (accscalar_t rand) {
        return static_cast<scalar_t>(median + sigma *
                __tanf(static_cast<accscalar_t>(M_PI) * (rand-static_cast<accscalar_t>(0.5))));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        cauchy_func);
    }
   });
}

void exponential_kernel(TensorIterator& iter, double lambda_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  // Note that HIP doesn't support std::nextafter in device code.
  auto nextafter_1_0_float = std::nextafter(1.0f, 0.0f);
  auto nextafter_1_0_double = std::nextafter(1.0, 0.0);
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "exponential_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto lambda = static_cast<accscalar_t>(lambda_);
    if (std::is_same<scalar_t, double>::value) {
      // define lambda for exponential transformation
      auto exponential_func = [lambda, nextafter_1_0_double] __device__ (accscalar_t rand) {
        if (lambda == static_cast<accscalar_t>(0.0)) {
          return static_cast<scalar_t>(0.0);
        }
        accscalar_t sample;
        // hiprand_uniform has (0,1] bounds. log(1) is 0 and exponential excludes 0.
        // Hence, squash the 1 to just below 1.
        if(rand == static_cast<accscalar_t>(1.0)) {
          sample = ::log(nextafter_1_0_double);
        } else {
          sample = ::log(rand);
        }
        return static_cast<scalar_t>(static_cast<accscalar_t>(-1.0) / lambda * sample);
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        exponential_func);
    } else {
      // use __logf fast approximation for peak bandwidth
      auto exponential_func = [lambda, nextafter_1_0_float] __device__ (accscalar_t rand) {
        if (lambda == static_cast<accscalar_t>(0.0)) {
          return static_cast<scalar_t>(0.0);
        }
        accscalar_t sample;
        if(rand == static_cast<accscalar_t>(1.0)) {
          sample = __logf(nextafter_1_0_float);
        } else {
          sample = __logf(rand);
        }
        return static_cast<scalar_t>(static_cast<accscalar_t>(-1.0) / lambda * sample);
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        exponential_func);
    }
   });
}

void geometric_kernel_cuda(TensorIterator& iter, double p_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "geometric_cuda", [&] {
    if (std::is_same<scalar_t, double>::value) {
      // define lambda for geometric transformation
      auto geometric_func = [p_] __device__ (double rand) {
        return static_cast<scalar_t>(::ceil(::log(rand) / ::log(static_cast<double>(1.0)-p_)));
      };
      distribution_nullary_kernel<scalar_t, double, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        geometric_func);
    } else {
      auto p = static_cast<float>(p_);
      auto geometric_func = [p] __device__ (float rand) {
        // use __logf fast approximation for peak bandwidth
        return static_cast<scalar_t>(::ceil(__logf(rand) / __logf(static_cast<float>(1.0)-p)));
      };
      distribution_nullary_kernel<scalar_t, float, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        geometric_func);
    }
   });
}

void log_normal_kernel(TensorIterator& iter, double mean_, double std_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "log_normal_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto mean = static_cast<accscalar_t>(mean_);
    auto std = static_cast<accscalar_t>(std_);
    if (std::is_same<scalar_t, double>::value) {
      // define lambda for log_normal transformation
      auto log_normal_func = [mean, std] __device__ (accscalar_t rand) {
        return static_cast<scalar_t>(::exp(rand * std + mean));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal2_double(state); },
        log_normal_func);
    } else {
      auto log_normal_func = [mean, std] __device__ (accscalar_t rand) {
        // use __expf fast approximation for peak bandwidth
        return static_cast<scalar_t>(__expf(rand * std + mean));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal4(state); },
        log_normal_func);
    }
   });
}

void bernoulli_scalar_cuda_kernel(TensorIterator& iter, double p_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_ALL_TYPES_AND3(
    at::ScalarType::Half, at::ScalarType::BFloat16, at::ScalarType::Bool, iter.dtype(), "bernoulli_scalar_cuda_", [&] {
      if (std::is_same<scalar_t, double>::value) {
      // define lambda for bernoulli transformation
      auto bernoulli_func = [p_] __device__ (double rand) {
        return static_cast<scalar_t>(rand <= p_);
      };
      distribution_nullary_kernel<scalar_t, double, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        bernoulli_func);
    } else {
      auto p = static_cast<float>(p_);
      auto bernoulli_func = [p] __device__ (float rand) {
        return static_cast<scalar_t>(rand <= p);
      };
      distribution_nullary_kernel<scalar_t, float, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        bernoulli_func);
    }
   });
}

Tensor& uniform_cuda_(Tensor& self, double from, double to, Generator* gen) {
  auto iter = TensorIterator::nullary_op(self);
  uniform_kernel_cuda(iter, from, to, gen);
  return self;
}

Tensor& normal_cuda_(Tensor& self, double mean, double std, Generator* gen) {
  TORCH_CHECK(std > 0.0, "normal_ expects std > 0.0, but found std=", std);
  auto iter = TensorIterator::nullary_op(self);
  normal_kernel_cuda(iter, mean, std, gen);
  return self;
}

Tensor& normal_out_cuda(Tensor& output, const Tensor& mean, double std, Generator* gen) {
  normal_cuda_(output, 0, std, gen);
  output.add_(mean);
  return output;
}

Tensor& normal_out_cuda(Tensor& output, double mean, const Tensor& std, Generator* gen) {
  normal_cuda_(output, 0, 1, gen);
  auto mean_tensor = at::full({}, mean, output.options());
  // NB: addcmul_out copies the tensor to be added into the output.
  // Please look at aten/src/THC/generic/THCTensorMathPointwise.cu
  // The previous function here was addcmul_out(output, mean_tensor, output, std, 1);
  // The third argument is not a constant reference and hence the samples in output are overwritten.
  // Consequently, the computation performed is mean_tensor + mean_tensor * std instead of mean_tensor + output * std
  output.mul_(std).add_(mean_tensor);
  return output;
}

Tensor& normal_out_cuda(Tensor& output, const Tensor& mean, const Tensor& std, Generator* gen) {
  bool is_deprecated_th_impl = resize_output_for_normal(output, mean, std);
  normal_cuda_(output, 0, 1, gen);
  // NB: addcmul_out copies the tensor to be added into the output.
  // Please look at aten/src/THC/generic/THCTensorMathPointwise.cu
  // The previous function here was addcmul_out(output, mean, output, std, 1);
  // The third argument is not a constant reference and hence the samples in output are overwritten.
  // Consequently, the computation performed is mean + mean * std instead of mean + output * std
  if (is_deprecated_th_impl) {
    output.mul_(std.reshape(mean.sizes())).add_(mean);
  }
  else {
    output.mul_(std).add_(mean);
  }
  return output;
}

Tensor normal_cuda(const Tensor& mean, double std, Generator* gen) {
  Tensor ret = at::empty_like(mean, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  normal_out_cuda(ret, mean, std, gen);
  return ret;
}

Tensor normal_cuda(double mean, const Tensor& std, Generator* gen) {
  Tensor ret = at::empty_like(std, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  normal_out_cuda(ret, mean, std, gen);
  return ret;
}

Tensor normal_cuda(const Tensor& mean, const Tensor& std, Generator* gen) {
  Tensor ret = at::empty({0}, mean.options(), LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  normal_out_cuda(ret, mean, std, gen);
  return ret;
}

Tensor& bernoulli_scalar_cuda_(Tensor &self, double p, Generator* gen) {
  TORCH_CHECK(0 <= p && p <= 1, "bernoulli_ expects p to be in [0, 1], but got p=", p);
  auto iter = TensorIterator::nullary_op(self);
  bernoulli_scalar_cuda_kernel(iter, p, gen);
  return self;
}

REGISTER_DISPATCH(cauchy_stub, &cauchy_kernel);
REGISTER_DISPATCH(exponential_stub, &exponential_kernel);
REGISTER_DISPATCH(geometric_stub, &geometric_kernel_cuda);
REGISTER_DISPATCH(log_normal_stub, &log_normal_kernel);
REGISTER_DISPATCH(random_from_to_stub, &random_from_to_kernel);
REGISTER_DISPATCH(random_stub, &random_kernel);
REGISTER_DISPATCH(random_full_64_bits_range_stub, &random_full_64_bits_range_kernel);

}} // namespace at::native
